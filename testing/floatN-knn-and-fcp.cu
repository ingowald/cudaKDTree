#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#define CUKD_ENABLE_STATS 1

#include "cukd/builder.h"
#include "cukd/spatial-kdtree.h"
// fcp = "find closest point" query
#include "cukd/fcp.h"
// knn = k-nearest-neighbor
#include "cukd/knn.h"
#include <queue>
#include <iomanip>
#include <random>

#if D_FROM_CMAKE == 2
using floatN = float2;
#elif D_FROM_CMAKE == 3
using floatN = float3;
#elif D_FROM_CMAKE == 4
using floatN = float4;
#elif D_FROM_CMAKE == 8
using floatN = cukd::vec_float<8>;
#else
#pragma error("error ... should get a value of 2, 3, or 4 from cmakefile...")
#endif

using namespace cukd;


#if EXPLICIT_DIM
struct PointAndDim {
  floatN point;
  int    dim;
};

struct PointAndDim_traits : public cukd::default_data_traits<floatN> {
  enum { has_explicit_dim = true };
  using data_t = PointAndDim;
  
  static inline __both__ const point_t &get_point(const PointAndDim &n) { return n.point; }
  static inline __both__ float get_coord(const PointAndDim &n, int d)
  { return cukd::get_coord(n.point,d); }
  
  static inline __both__ int  get_dim(const PointAndDim &p) 
  { return p.dim; }
	   
  static inline __both__ void set_dim(PointAndDim &p, int dim) 
  { p.dim = dim; }
};

using data_t = PointAndDim;
using data_traits = PointAndDim_traits;
#else
using data_t = floatN;
using data_traits = default_data_traits<floatN>;
#endif


floatN *generatePoints(int N)
{
  static int g_seed = 100000;
  std::seed_seq seq{g_seed++};
  // std::random_device rd(seq());  // Will be used to obtain a seed for the random number engine
  std::default_random_engine rd(seq);
  std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
  std::uniform_int_distribution<> dist(0,N);

  std::cout << "generating " << N << " uniform random points" << std::endl;
  floatN *d_points = 0;
  hipMallocManaged((char **)&d_points,N*sizeof(*d_points));
  if (!d_points)
    throw std::runtime_error("could not allocate points mem...");
  
  enum { num_dims = num_dims_of<floatN>::value };
  for (int i=0;i<N;i++)
    for (int d=0;d<num_dims;d++) {
      ((float *)&d_points[i])[d] = (float)dist(gen);
    }
  return d_points;
}



__global__
void d_fcp(float   *d_results,
#if SPATIAL
           SpatialKDTree<data_t,data_traits> tree,
#endif
           floatN  *d_queries,
           int      numQueries,
           const cukd::box_t<floatN> *d_bounds,
           data_t  *d_nodes,
           int      numNodes,
           float    cutOffRadius)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numQueries) return;

  using point_t = floatN;
  point_t queryPos = d_queries[tid];
  FcpSearchParams params;
  params.cutOffRadius = cutOffRadius;
#if SPATIAL
  int closestID
    = TRAVERSAL_METHOD::fcp
    <data_t,data_traits>
    (tree,queryPos,params);
#else
  int closestID
    = TRAVERSAL_METHOD::fcp
    <data_t,data_traits>
    (queryPos,
     *d_bounds,
     d_nodes,numNodes,params);
#endif
  
  d_results[tid]
    = (closestID < 0)
    ? INFINITY
    : distance(queryPos,data_traits::get_point(d_nodes[closestID]));
}




template<typename CandidateList>
__global__
void d_knn(float   *d_results,
#if SPATIAL
           SpatialKDTree<data_t,data_traits> tree,
#endif
           floatN  *d_queries,
           int      numQueries,
           const cukd::box_t<floatN> *d_bounds,
           data_t  *d_nodes,
           int      numNodes,
           float    cutOffRadius)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numQueries) return;
  
  CandidateList result(cutOffRadius);
#if SPATIAL
  float sqrDist
    = cukd::TRAVERSAL_METHOD::knn<CandidateList,data_t,data_traits>
    (result,
     tree,
     d_queries[tid]);
#else
  float sqrDist
    = cukd::TRAVERSAL_METHOD::knn<CandidateList,data_t,data_traits>
    (result,
     d_queries[tid],
     *d_bounds,
     d_nodes,numNodes);
#endif
  d_results[tid] = sqrtf(sqrDist);
}



void run_kernel(float  *d_results,
                floatN *d_queries,
                int     numQueries,
#if SPATIAL
                SpatialKDTree<floatN> &tree,
#endif
                const cukd::box_t<floatN> *d_bounds,
                data_t *d_nodes,
                int     numNodes,
#if USE_KNN
                int k,
#endif
                float   cutOffRadius
                )
{
  int bs = 128;
  int nb = divRoundUp(numQueries,bs);
  unsigned long long *d_stats = 0;
  static bool firstTime = true;
  CUKD_CUDA_SYNC_CHECK();
  CUKD_STATS(if (firstTime) {
      hipMallocManaged((char **)&d_stats,sizeof(*d_stats));
      *d_stats = 0;
      void *symAddr = 0;
      hipGetSymbolAddress(&symAddr,HIP_SYMBOL(cukd::g_traversalStats));
      CUKD_STATS(hipMemcpy(symAddr,&d_stats,sizeof(d_stats),hipMemcpyHostToDevice));
    }
    )
  CUKD_CUDA_SYNC_CHECK();
  
#if USE_KNN
  if (k == 4)
    d_knn<FixedCandidateList<4>><<<nb,bs>>>
      (d_results,
#if SPATIAL
       tree,
#endif
       d_queries,numQueries,
       d_bounds,
       d_nodes,numNodes,cutOffRadius);
  else if (k == 8)
    d_knn<FixedCandidateList<8>><<<nb,bs>>>
      (d_results,
#if SPATIAL
       tree,
#endif
       d_queries,numQueries,
       d_bounds,
       d_nodes,numNodes,cutOffRadius);
  else if (k == 64)
    d_knn<HeapCandidateList<64>><<<nb,bs>>>
      (d_results,
#if SPATIAL
       tree,
#endif
       d_queries,numQueries,
       d_bounds,
       d_nodes,numNodes,cutOffRadius);
  else if (k == 20)
    d_knn<HeapCandidateList<20>><<<nb,bs>>>
      (d_results,
#if SPATIAL
       tree,
#endif
       d_queries,numQueries,
       d_bounds,
       d_nodes,numNodes,cutOffRadius);
  else if (k == 50)
    d_knn<HeapCandidateList<50>><<<nb,bs>>>
      (d_results,
#if SPATIAL
       tree,
#endif
       d_queries,numQueries,
       d_bounds,
       d_nodes,numNodes,cutOffRadius);
  else
    throw std::runtime_error("unsupported k for knn queries");
#else
  d_fcp<<<nb,bs>>>
    (d_results,
#if SPATIAL
     tree,
#endif
     d_queries,numQueries,
     d_bounds,
     d_nodes,numNodes,cutOffRadius);
#endif
  CUKD_CUDA_SYNC_CHECK();
  if (firstTime) {
    hipDeviceSynchronize();
    CUKD_STATS(
               double checkSum = 0.;
               for (int i=0;i<numQueries;i++)
                 checkSum += d_results[i];
               std::cout << "CHECKSUM " << checkSum << std::endl;
               std::cout << "KDTREE_STATS " << *d_stats << " CHECKSUM " << checkSum << std::endl;
               std::cout << "NICE_STATS " << common::prettyNumber(*d_stats) << std::endl;
               );
    hipFree(d_stats);
    firstTime = false;
  }
  CUKD_CUDA_SYNC_CHECK();
}

#if EXPLICIT_DIM
__global__ void copyPoints(PointAndDim *d_points,
                           floatN *d_inputs,
                           int numPoints)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numPoints) return;
  d_points[tid].point = d_inputs[tid];
}
#endif

template<typename data_t, typename data_traits>
void verifyKNN(int pointID,
               int k,
               float maxRadius,
               data_t *points, int numPoints,
               floatN queryPoint,
               float reportedResult)
{
  using point_t = typename data_traits::point_t;
  std::priority_queue<float> closest_k;
  for (int i=0;i<numPoints;i++) {
    point_t point_i = data_traits::get_point(points[i]);
    float d = sqrDistance(queryPoint,point_i);
    if (d >= maxRadius*maxRadius)
      continue;
    
    closest_k.push(d);
    if (closest_k.size() > k)
      closest_k.pop();
  }
  
  float actualResult = (closest_k.size() == k) ? sqrtf(closest_k.top()) : maxRadius;
  
  // check if the top 21-ish bits are the same; this will allow the
  // compiler to produce slightly different results on host and device
  // (usually caused by it uses madd's on one and separate +/* on
  // t'other...
  bool closeEnough
    =  /* this catches result==inf:*/
    (actualResult == reportedResult)
    || /* this catches bit errors: */
    (fabsf(actualResult - reportedResult)/std::max(actualResult,reportedResult) <= 1e-6f);
  
  if (!closeEnough) {
    std::cout << "for point #" << pointID << ": "
              << "verify found max dist " << std::setprecision(10) << actualResult
              << " (bits " << (int*)(uint64_t)((uint32_t&)actualResult)
              << "), knn reported " << reportedResult
              << " (bits " << (int*)(uint64_t)((uint32_t&)reportedResult)
              << "), difference is " << (actualResult-reportedResult)
              << std::endl;
    throw std::runtime_error("verification failed");
  }
}

template<typename data_t, typename data_traits>
void checkRec(data_t *nodes, int numNodes,
              const cukd::box_t<typename data_traits::point_t> &bounds,
              int curr)
{
  using point_t  = typename data_traits::point_t;
  using scalar_t = typename scalar_type_of<point_t>::type;
  enum { num_dims = num_dims_of<point_t>::value };
    
  if (curr >= numNodes) return;

  point_t point = data_traits::get_point(nodes[curr]);
  
  if (!bounds.contains(point))
    throw std::runtime_error
      ("invalid k-d tree - node "+std::to_string(curr)+" not in parent bounds");
  
  const int  curr_dim
    = data_traits::has_explicit_dim
    ? data_traits::get_dim(nodes[curr])
    : (BinaryTree::levelOf(curr) % num_dims);

  const scalar_t curr_s = data_traits::get_coord(nodes[curr],curr_dim);
  
  cukd::box_t<point_t> lBounds = bounds;
  get_coord(lBounds.upper,curr_dim) = curr_s;
  cukd::box_t<point_t> rBounds = bounds;
  get_coord(rBounds.lower,curr_dim) = curr_s;

  checkRec<data_t,data_traits>(nodes,numNodes,lBounds,2*curr+1);
  checkRec<data_t,data_traits>(nodes,numNodes,rBounds,2*curr+2);
}

template<typename data_t, typename data_traits>
void checkTree(data_t *nodes, int numNodes, std::vector<data_t> &savedNodes)
{
  cukd::box_t<floatN> bounds;
  bounds.setInfinite();
  checkRec<data_t,data_traits>(nodes,numNodes,bounds,0);
  std::cout << "** verify: tree checked, and valid k-d tree" << std::endl;
}




template<typename data_t, typename data_traits>
void checkRec(SpatialKDTree<data_t,data_traits> &tree,
              const cukd::box_t<typename data_traits::point_t> &bounds,
              int nodeID)
{
  using point_t  = typename data_traits::point_t;
  using scalar_t = typename scalar_type_of<point_t>::type;
  enum { num_dims = num_dims_of<point_t>::value };

  auto &node = tree.nodes[nodeID];
  if (node.count > 0) {
    for (int i=0;i<node.count;i++) {
      int primID = tree.primIDs[node.offset+i];
      point_t point = data_traits::get_point(tree.data[primID]);
      if (!bounds.contains(point))
        throw std::runtime_error
          ("invalid k-d tree - prim "+std::to_string(primID)+" not in parent bounds");
    }
    return;
  }
  
  const scalar_t curr_s = node.pos;
  
  cukd::box_t<point_t> lBounds = bounds;
  set_coord(lBounds.upper,node.dim,curr_s);
  cukd::box_t<point_t> rBounds = bounds;
  set_coord(rBounds.lower,node.dim,curr_s);

  checkRec<data_t,data_traits>(tree,lBounds,node.offset+0);
  checkRec<data_t,data_traits>(tree,rBounds,node.offset+1);
}

template<typename data_t, typename data_traits>
void checkTree(SpatialKDTree<data_t,data_traits> &tree)
{
  cukd::box_t<floatN> bounds = tree.bounds;
  checkRec<data_t,data_traits>(tree,bounds,0);
  std::cout << "** verify: tree checked, and valid spatial-k-d tree" << std::endl;
}

template<typename data_t, typename data_traits>
void verifyFCP(int pointID,
               float cutOffRadius,
               data_t *points, int numPoints,
               floatN queryPoint,
               float reportedResult)
{
  using point_t = typename data_traits::point_t;
  float actualResult = INFINITY;
  for (int i=0;i<numPoints;i++) {
    point_t point_i = data_traits::get_point(points[i]);
    float d = sqrDistance(queryPoint,point_i);
    if (d >= cutOffRadius*cutOffRadius)
      continue;

    actualResult = std::min(actualResult,sqrtf(d));
  }
  
  
  // check if the top 21-ish bits are the same; this will allow the
  // compiler to produce slightly different results on host and device
  // (usually caused by it using madd's on one and separate +/* on
  // t'other...
  bool closeEnough
    =  /* this catches result==inf:*/
    (actualResult == reportedResult)
    || /* this catches bit errors: */
    (fabsf(actualResult - reportedResult)/std::max(actualResult,reportedResult) <= 1e-6f);
  
  if (!closeEnough) {
    std::cout << "for point #" << pointID << ": "
              << "verify found max dist " << std::setprecision(10) << actualResult
              << " (bits " << (int*)(uint64_t)((uint32_t&)actualResult)
              << "), knn reported " << reportedResult
              << " (bits " << (int*)(uint64_t)((uint32_t&)reportedResult)
              << "), difference is " << (actualResult-reportedResult)
              << std::endl;
    throw std::runtime_error("verification failed");
  }
}


int main(int ac, const char **av)
{
  using namespace cukd::common;

  int    numPoints = 1000000;
  bool   verify = false;
  int    nRepeats = 1;
  size_t numQueries = 1000000;
  float  cutOffRadius = std::numeric_limits<float>::infinity();
#if SPATIAL
  cukd::BuildConfig buildConfig = {};
#endif
#if USE_KNN
  int    k = 50;
#endif
  for (int i=1;i<ac;i++) {
    std::string arg = av[i];
    if (arg[0] != '-')
      numPoints = std::stoi(arg);
    else if (arg == "-v")
      verify = true;
    else if (arg == "-nq")
      numQueries = atoi(av[++i]);
    else if (arg == "-nr")
      nRepeats = atoi(av[++i]);
#if SPATIAL
    else if (arg == "-lt")
      buildConfig.makeLeafThreshold = std::stoi(av[++i]);
#endif
    else if (arg == "-r")
      cutOffRadius = std::stof(av[++i]);
    else if (arg == "--load-dumped-files")
    {
        numPoints = 0;  numQueries = 0;
    }
#if USE_KNN
    else if (arg == "-k")
      k = std::stoi(av[++i]);
#endif
    else
      throw std::runtime_error("known cmdline arg "+arg);
  }
  
  floatN *d_inputs
    = numPoints
    ? generatePoints(numPoints)
    : loadPoints<floatN>("data_points",numPoints);
#if EXPLICIT_DIM
  PointAndDim *d_points;
  hipMallocManaged((void**)&d_points,numPoints*sizeof(*d_points));
  copyPoints<<<divRoundUp(numPoints,128),128>>>
    (d_points,d_inputs,numPoints);
  using data_t = PointAndDim;
#else
  floatN *d_points = d_inputs;
  using data_t = floatN;
#endif
  
  cukd::box_t<floatN> *d_bounds;
  hipMallocManaged((void**)&d_bounds,sizeof(cukd::box_t<floatN>));
  std::cout << "allocated memory for the world space bounding box ..." << std::endl;
#if SPATIAL
  SpatialKDTree<data_t,data_traits> tree;
#endif
  {
    std::vector<data_t> saved_points;
    if (verify) {
      saved_points.resize(numPoints);
      std::copy(d_points,d_points+numPoints,saved_points.data());
    }
    std::cout << "calling builder..." << std::endl;
    double t0 = getCurrentTime();
#if SPATIAL 
    cukd::buildTree<data_t,data_traits>
      (tree,d_points,numPoints,buildConfig);
#else
    cukd::buildTree<data_t,data_traits>
      (d_points,numPoints,d_bounds);
#endif
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done building tree, took "
              << prettyDouble(t1-t0) << "s" << std::endl;

#if SPATIAL
    if (verify)
      checkTree<data_t,data_traits>(tree);
#else
    if (verify)
      checkTree<data_t,data_traits>(d_points,numPoints,saved_points);
#endif
  }
  
  floatN *d_queries
    = numQueries
    ? generatePoints((int)numQueries)
    : loadPoints<floatN>("query_points",numQueries);
  float  *d_results;
  CUKD_CUDA_CALL(MallocManaged((void**)&d_results,numQueries*sizeof(*d_results)));
  CUKD_CUDA_SYNC_CHECK();
  {
    double t0 = getCurrentTime();
    for (int i=0;i<nRepeats;i++) {
      run_kernel
        (d_results,d_queries,(int)numQueries,
#if SPATIAL
         tree,
#endif
         d_bounds,
         d_points,numPoints,
#if USE_KNN
         k,
#endif
         cutOffRadius);
    }
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done " << nRepeats
              << " iterations of " << numQueries
              << " fcp queries, took " << prettyDouble(t1-t0)
              << "s" << std::endl;
    std::cout << "that is " << prettyDouble(numQueries*nRepeats/(t1-t0))
              << " queries/s" << std::endl;
  }

  double checkSum = 0.;
  for (int i=0;i<numQueries;i++)
    checkSum += d_results[i];
  std::cout << "CHECKSUM " << checkSum << std::endl;
  
  if (verify) {
    std::cout << "verifying ..." << std::endl;
    for (int i=0;i<numQueries;i++) {
      floatN qp           = d_queries[i];
      float  reportedResult = d_results[i];
#if USE_KNN
      verifyKNN<data_t,data_traits>
        (i,k,cutOffRadius,d_points,numPoints,qp,reportedResult);
#else
      verifyFCP<data_t,data_traits>
        (i,cutOffRadius,d_points,numPoints,qp,reportedResult);
#endif          
    }
    std::cout << "verification succeeded... done." << std::endl;
  }
}
  
