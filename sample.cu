#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "cukd/builder.h"
// fcp = "find closest point" query
#include "cukd/fcp.h"
#include <queue>
#include <iomanip>
#include <random>

using namespace cukd;

float3 *generatePoints(int N)
{
  static int g_seed = 100000;
  std::seed_seq seq{g_seed++};
  std::default_random_engine rd(seq);
  std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
  std::uniform_int_distribution<> dist(0,N);

  std::cout << "generating " << N << " uniform random points" << std::endl;
  float3 *d_points = 0;
  hipMallocManaged((char **)&d_points,N*sizeof(*d_points));
  if (!d_points)
    throw std::runtime_error("could not allocate points mem...");
  
  for (int i=0;i<N;i++) {
    d_points[i].x = (float)dist(gen);
    d_points[i].y = (float)dist(gen);
    d_points[i].z = (float)dist(gen);
  }
  return d_points;
}


__global__
void d_fcp(float   *d_results,
           float3  *d_queries,
           int      numQueries,
           /*! the world bounding box computed by the builder */
           const cukd::box_t<float3> *d_bounds,
           float3  *d_nodes,
           int      numNodes,
           float    cutOffRadius)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numQueries) return;

  using point_t = float3;
  point_t queryPos = d_queries[tid];
  FcpSearchParams params;
  params.cutOffRadius = cutOffRadius;
  int closestID
    = cukd::cct::fcp
    (queryPos,*d_bounds,d_nodes,numNodes,params);
  
  d_results[tid]
    = (closestID < 0)
    ? INFINITY
    : distance(queryPos,d_nodes[closestID]);
}




int main(int ac, const char **av)
{
  using namespace cukd::common;

  int    numPoints = 1000000;
  int    nRepeats = 1;
  size_t numQueries = 1000000;
  float  cutOffRadius = std::numeric_limits<float>::infinity();
  for (int i=1;i<ac;i++) {
    std::string arg = av[i];
    if (arg[0] != '-')
      numPoints = std::stoi(arg);
    else if (arg == "-nq")
      numQueries = atoi(av[++i]);
    else if (arg == "-nr")
      nRepeats = atoi(av[++i]);
    else if (arg == "-r")
      cutOffRadius = std::stof(av[++i]);
    else
      throw std::runtime_error("known cmdline arg "+arg);
  }
  
  // ==================================================================
  // create sample input point that we'll build the tree over
  // ==================================================================
  float3 *d_points = generatePoints(numPoints);

  // ==================================================================
  // allocate some memory for the world-space bounding box, so the
  // builder can compute and return that for our chosen traversal
  // method to use
  // ==================================================================
  cukd::box_t<float3> *d_bounds;
  hipMallocManaged((void**)&d_bounds,sizeof(cukd::box_t<float3>));
  std::cout << "allocated memory for the world space bounding box ..." << std::endl;

  // ==================================================================
  // build the tree. this will also comptue the world-space boudig box
  // of all points
  // ==================================================================
  std::cout << "calling builder..." << std::endl;
  double t0 = getCurrentTime();
  cukd::buildTree(d_points,numPoints,d_bounds);
  CUKD_CUDA_SYNC_CHECK();
  double t1 = getCurrentTime();
  std::cout << "done building tree, took "
            << prettyDouble(t1-t0) << "s" << std::endl;

  // ==================================================================
  // create set of sample query points
  // ==================================================================
  float3 *d_queries
    = generatePoints(numQueries);
  // allocate memory for the results
  float  *d_results;
  CUKD_CUDA_CALL(MallocManaged((void**)&d_results,numQueries*sizeof(*d_results)));


  // ==================================================================
  // and do some queryies - let's do the same ones in a loop so we cna
  // measure perf.
  // ==================================================================
  {
    double t0 = getCurrentTime();
    for (int i=0;i<nRepeats;i++) {
      int bs = 128;
      int nb = divRoundUp((int)numQueries,bs);
      d_fcp<<<nb,bs>>>
        (d_results,d_queries,numQueries,
         d_bounds,d_points,numPoints,cutOffRadius);
      hipDeviceSynchronize();
    }
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done " << nRepeats
              << " iterations of " << numQueries
              << " fcp queries, took " << prettyDouble(t1-t0)
              << "s" << std::endl;
    std::cout << "that is " << prettyDouble(numQueries*nRepeats/(t1-t0))
              << " queries/s" << std::endl;
  }
  
}
  
